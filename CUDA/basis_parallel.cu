#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>


struct timeval t_s, t_e, t_s2, t_e2, t_s3, t_e3;
double t1, t2, t3;

__global__ void compute_Dontcares(char* image,int* row_flag,int* col_flag,char* result, int i, int j, int image_width)
{
	int row = (blockIdx.x + i);
	int col = threadIdx.x + j;
	if(image[row*image_width+col] == image[(row-i)*image_width+col-j])
	{
		if(image[row*image_width+col] == '1')
			result[(row-i)*(image_width-j)+ (col-j)]='1';

		else//(image[row*image_width+col] == '0')
			result[(row-i)*(image_width-j)+ (col-j)]='0';

		row_flag[row-i]=1;
		col_flag[col-j]=1;
	}
	else
		result[(row-i)*(image_width-j)+ (col-j)]='o';
}


__global__ void compute_Result(char* ptr_ptr ,int image_height, int image_width, int row_start,int col_start, int col_end, char* result2_ptr, int j)    {
	int row = blockIdx.x + row_start;
	int col = threadIdx.x + col_start;
	result2_ptr[(row - row_start) * (col_end - col_start + 1) + (col - col_start)] = ptr_ptr[row*(image_width-j) + col];
}

__global__
void compute_count(int image_width, char* result, int* occurance_count, char* image, tuple* temp_occurance_list, int height, int width)
{
	int row_offset = threadIdx.x;
	for(int col_offset=0; col_offset<=image_width - width; col_offset++)
	{

		int count = 0;
		for(int row = 0; row<height; row++)
		{
			for(int col = 0; col<width; col++)
			{
				if(result[row*width + col] == 'o' )
				{
					count++;
				}
				else
				{
					if(result[row*width + col] == image[(row+row_offset)*image_width + (col+col_offset)])
					{
						count++;
					}
					else
						break;
				}
			}
		}
		
		if(count == width*height)
		{
			tuple t;
			t.i = row_offset;
			t.j = col_offset;

			int old_count = atomicInc((unsigned int*)(&(occurance_count[0])),(unsigned int)999999999);
			temp_occurance_list[old_count] = t;
		}
	}
}

ConsensusGrid consensus_parallel(int i, int j,char* image, int image_height, int image_width)
{
	ConsensusGrid consensus_grid;
	char* image_ptr;
	int* row_flag_ptr;
	int*  col_flag_ptr;
	char * ptr_ptr;
	char*  result2_ptr;
	char* result_ptr;

	char *result = (char*)malloc(sizeof(char)*(image_height-i)*(image_width-j));
	char* ptr = result;

	int* row_flag=(int*)calloc(sizeof(int),(image_height-i));
	int* col_flag=(int*)calloc(sizeof(int),(image_width-j));

	//copy result and image to gpu
	//compute the result array
	//copy result back to cpu
	hipMalloc((void**)&image_ptr, sizeof(char)*(image_height*image_width));
	hipMemcpy(image_ptr, image, sizeof(char)*(image_height*image_width) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&row_flag_ptr, sizeof(int)*(image_height-i));
	hipMemcpy(row_flag_ptr, row_flag, sizeof(int)*(image_height-i) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&col_flag_ptr, sizeof(int)*(image_width-j));
	hipMemcpy(col_flag_ptr, col_flag, sizeof(int)*(image_width-j) ,  hipMemcpyHostToDevice);
	hipMalloc((void**)&result_ptr, sizeof(char)*(image_width-j)*(image_height-i));

	gettimeofday(&t_s, NULL);

	// __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
	compute_Dontcares<<<(image_height-i),(image_width-j)>>>(image_ptr,row_flag_ptr, col_flag_ptr, result_ptr,i,j, image_width);

	hipDeviceSynchronize();

	gettimeofday(&t_e, NULL);
	t1 = (((double)t_e.tv_sec-(double)t_s.tv_sec)*1000) + ((double)t_e.tv_usec - (double)t_s.tv_usec)/1000;

	hipMemcpy(result, result_ptr, sizeof(char)*(image_height-i)*(image_width-j),  hipMemcpyDeviceToHost);
	hipMemcpy(row_flag, row_flag_ptr, sizeof(int)*(image_height-i),  hipMemcpyDeviceToHost);
	hipMemcpy(col_flag, col_flag_ptr, sizeof(int)*(image_width-j),  hipMemcpyDeviceToHost);
	hipMemcpy(image, image_ptr, sizeof(char)*(image_height)*(image_width),  hipMemcpyDeviceToHost);
	
	int col_start,col_end, row_start, row_end;
	for(row_start=0; row_start<(image_height-i); row_start++)
	{
		if(row_flag[row_start]==1)
			break;
	}
	for(row_end=(image_height-i-1); row_end>=0; row_end--)
	{
		if(row_flag[row_end]==1)
			break;
	}
	for(col_start=0; col_start<(image_width-j); col_start++)
	{
		if(col_flag[col_start]==1)
			break;
	}
	for(col_end=(image_width-j-1); col_end>=0; col_end--)
	{
		if(col_flag[col_end]==1)
			break;
	}

	if((row_start > row_end) || (col_start > col_end) )
	{
		consensus_grid.result = NULL;
		consensus_grid.height = 0;
		consensus_grid.width = 0;
		return consensus_grid;
	}

	char* result2 = (char*)malloc(sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1));
	hipMalloc((void**)&result2_ptr, sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1));

	hipMalloc((void**)&ptr_ptr, sizeof(char)*(image_height-i)*(image_width-j));
	hipMemcpy(ptr_ptr, ptr, sizeof(char)*(image_height-i)*(image_width-j) ,  hipMemcpyHostToDevice);

	consensus_grid.result = result2;

	gettimeofday(&t_s2, NULL);

	compute_Result<<<(row_end-row_start+1),(col_end-col_start+1)>>>(ptr_ptr , image_height, image_width, row_start,col_start, col_end, result2_ptr, j);

	hipDeviceSynchronize();

	gettimeofday(&t_e2, NULL);
	t2 = (((double)t_e2.tv_sec-(double)t_s2.tv_sec)*1000) + ((double)t_e2.tv_usec - (double)t_s2.tv_usec)/1000;

	hipMemcpy(result2, result2_ptr, sizeof(char)*(row_end-row_start+1)*(col_end-col_start+1),  hipMemcpyDeviceToHost);


	consensus_grid.height = row_end - row_start + 1;
	consensus_grid.width = col_end - col_start + 1;
	consensus_grid.occurance = NULL;
	consensus_grid.occurance_count = 0;

	free(ptr);
	free(row_flag);
	free(col_flag);
	return consensus_grid;
}

void calculate_list_parallel()
{
	char* image_ptr;
	tuple* temp_occurance_list_ptr;
	char* result_ptr;
	int* occurance_count_ptr;
	pthread_mutex_t lock;
	pthread_mutex_init(&lock,NULL);

	for(int i=0; i<image_height*image_width; i++)
	{

		if((meet[i].height != 0) && (meet[i].width != 0))
		{

			tuple* temp_occurance_list = new tuple[image_height*image_width];

			hipMalloc((void**)&image_ptr, sizeof(char)*image_height*image_width);
			hipMemcpy(image_ptr, image, sizeof(char)*image_height*image_width, hipMemcpyHostToDevice);
			hipMalloc((void**)&temp_occurance_list_ptr, sizeof(tuple)*image_height*image_width);
			hipMemcpy(temp_occurance_list_ptr, temp_occurance_list, sizeof(tuple)*image_height*image_width, hipMemcpyHostToDevice);

			hipMalloc((void**)&result_ptr, sizeof(char)*meet[i].height*meet[i].width);
                        hipMemcpy(result_ptr, meet[i].result, sizeof(char)*meet[i].height*meet[i].width, hipMemcpyHostToDevice);
			
			hipMalloc((void**)&occurance_count_ptr, sizeof(int));
                        
			hipMemcpy(occurance_count_ptr, &(meet[i].occurance_count), sizeof(int), hipMemcpyHostToDevice);

			gettimeofday(&t_s3, NULL);

			compute_count<<< 1, image_height-meet[i].height+1>>> (image_width,result_ptr, occurance_count_ptr, image_ptr, temp_occurance_list_ptr, meet[i].height, meet[i].width);

			hipDeviceSynchronize();

			gettimeofday(&t_e3, NULL);
			t3 = (((double)t_e3.tv_sec-(double)t_s3.tv_sec)*1000) + ((double)t_e3.tv_usec - (double)t_s3.tv_usec)/1000;


			hipMemcpy(temp_occurance_list, temp_occurance_list_ptr, sizeof(tuple)*image_height*image_width, hipMemcpyDeviceToHost);
			hipMemcpy(&meet[i].occurance_count,occurance_count_ptr, sizeof(int), hipMemcpyDeviceToHost);

			if(meet[i].occurance_count >=0)
			{
				meet[i].occurance = new tuple[meet[i].occurance_count];
				for(int x=0; x<meet[i].occurance_count; x++)
				{
					meet[i].occurance[x] = temp_occurance_list[x];
				}

				not_null_count++;
			}
			else
			{
				meet[i].occurance_count = 0;
			}

			free(temp_occurance_list);
hipDeviceReset();
		}
	}
}

